#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <string.h>

#ifndef __HIPCC__
#define __HIPCC__
#endif // !__HIPCC__


__global__ void GAUSS(double* mat, int N, double* det)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    double del;
    if (x < N)
    {
        int check = 1;
        for (int i = 0; i < N - 1; i++) // идем по столбцам матрицы
        {
             // каждый поток ждет, пока остальные потоки блока достигнут этой точки
            if (x > i)
            {
                del = mat[x * N + i] / mat[i * N + i];
                for (int j = i; j < N; j++)
                    mat[x * N + j] -= del * mat[i * N + j];
            }
            else
                break;
        }
        det[x] = mat[(x)*N + x];
        /*for (int i = 0; i < N; i++)
            printf("%f ",det[i]);
        printf("\n%d\n", x);*/
        __syncthreads();
        if (x == N - 1)
        {
            for (int i = 1; i < N; i++)
                det[0] *= det[i];
            det[0] *= check;
        }
    }
}


void print_matrix(int N, double* matrix)
{
    int k = 10;
    printf("┌");
    for (int i = 1; i < N * k; i++)
    {
        if (!(i % k))
            printf("┬");
        else
            printf("─");
    }
    printf("┐\n");

    for (int y = 0; y < N; y++)
    {
        for (int x = 0; x < N; x++)
        {
            printf("│ %7.3f ", matrix[y * N + x]);
        }
        printf("│\n");
        if (y != N - 1)
        {
            printf("├");
            for (int i = 1; i < N * k; i++)
            {
                if (!(i % k))
                    printf("┼");
                else
                    printf("─");
            }
            printf("┤\n");
        }
    }

    printf("└");
    for (int i = 1; i < N * k; i++)
    {
        if (!(i % k))
            printf("┴");
        else
            printf("─");
    }
    printf("┘\n");
}

double determinantCPU(int N, double* matrix)
{

    double det = 1, temp, coefficient;
    for (int i = 0; i < N - 1; i++)
    {
        temp = matrix[i * N + i];       // элемент на строчке, ниже которого будут нули
        for (int j = i + 1; j < N; j++) // идем вниз по строчкам и обнуляем элементы столбца, остальные элементы строки мы вычитаем
        {
            coefficient = matrix[j * N + i] / temp; // коэффициент, на который умножаются элементы вверхней строки для зануления
            for (int s = i; s < N; s++)
                matrix[j * N + s] -= coefficient * matrix[i * N + s]; // идём по строке и из текущего элемента вычитаем (k*элемент сверху)
        }
    }
    for (int i = 0; i < N; i++)
        det *= matrix[i * N + i]; // определитель полученной матрицы равен произведению элементов на главной диагонали
    return det;
}

int main()
{
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    // printf("Multiprocessor count: %d\n", deviceProp.multiProcessorCount);
    int N;
    printf("Matrix NxN \nEnter N: ");
    scanf("%d", &N);
    int blockSize;
    printf("Enter blockSize: ");
    scanf("%d", &blockSize);
    double* matrixHost;
    double* matrix;
    matrixHost = (double*)malloc(sizeof(double) * N * N);
    matrix = (double*)malloc(sizeof(double) * N * N);
    // инициализация матрицы
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
            matrixHost[i * N + j] = 1 + rand() % 5;
    }
    //print_matrix(N, matrixHost);
    memcpy(matrix, matrixHost, sizeof(double) * N * N);
    // определитель на CPU
    float start = clock();
    double determ = determinantCPU(N, matrixHost);
    float end = clock();

    free(matrixHost);

    printf("CPU determinant = %.2f\n", determ);
    printf("Time CPU: %f\n\n", (end - start) * 1000.0 / CLOCKS_PER_SEC);

    // определитель на GPU
    double determinant = 1;
    double* matrixDevice = 0;
    double* determinant2;
    // Выделяем память для данных, которые будут на GPU
    hipMalloc((void**)&matrixDevice, (unsigned long long) (N * N * sizeof(double)));
    hipMalloc((void**)&determinant2, N * sizeof(double));
    // Передаем в matrixDevice значения matrix
    // копирование исходной матрицы в память GPU
    hipMemcpy(matrixDevice, matrix, N * N * sizeof(double), hipMemcpyHostToDevice);
    // Запускаем ядро
    start = clock();
    // Запуск ядра из blockSize блока по 1024 потоков
    GAUSS <<<blockSize, 1024 >>> (matrixDevice, N, determinant2);
    hipDeviceSynchronize();
    // передаем значение определителя обратно в CPU
    hipMemcpy(&determinant, &(determinant2[0]), sizeof(double), hipMemcpyDeviceToHost);
    end = clock();
    printf("GPU determinant = %.2f\n", determinant);
    printf("Time GPU: %f\n", (end - start) * 1000.0 / CLOCKS_PER_SEC);

    hipFree(matrixDevice);
    hipFree(determinant2);
    free(matrix);
}