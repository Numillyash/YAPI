#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void add(int a, int b, int *c)
{
    *c = a + b;
}

int main()
{
    int c;
    int *dev_c;

    hipMalloc((void **)&dev_c, sizeof(int));
    add<<<1, 1>>>(1, 2, dev_c);
    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
    printf("%i\n", c);
    return 0;
}